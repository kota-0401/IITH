#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(int *dA, int *dB, int *dC, int n) {
    int sum = 0;
    for (int k = 0; k < n; k++) {
        sum += (dA[threadIdx.x * n + k] * dB[k * blockDim.y + threadIdx.y]);
    }
    dC[threadIdx.x * blockDim.y + threadIdx.y] = sum;
}

void read_matrix(FILE* file, int *A, int *B, int m, int n, int l) {
    // Read Matrix A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            fscanf(file, "%d", &A[i * n + j]);
        }
    }

    // Read Matrix B
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < l; j++) {
            fscanf(file, "%d", &B[i * l + j]);
        }
    }
}

int main() {
    FILE* input_file = fopen("input.txt", "r");
    if (input_file == NULL) {
        printf("Error opening input.txt\n");
        return 1;
    }

    int m, n, l;
    fscanf(input_file, "%d %d", &m, &n);
    fscanf(input_file, "%d %d", &n, &l);

    int *A = (int *)malloc(m * n * sizeof(int));
    int *B = (int *)malloc(n * l * sizeof(int));
    int *C = (int *)malloc(m * l * sizeof(int));

    read_matrix(input_file, A, B, m, n, l);
    fclose(input_file);

    int *dA, *dB, *dC;
    hipMalloc(&dA, m * n * sizeof(int));
    hipMalloc(&dB, n * l * sizeof(int));
    hipMalloc(&dC, m * l * sizeof(int));

    // Memory copy from host to device
    hipMemcpy(dA, A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * l * sizeof(int), hipMemcpyHostToDevice);

    free(A);
    free(B);

    dim3 block(m, l, 1);
    
    // Launching the GPU kernel
    kernel<<<1, block>>>(dA, dB, dC, n);

    // Memory copy from device to host (Also Synchronizing)
    hipMemcpy(C, dC, m * l * sizeof(int), hipMemcpyDeviceToHost);

    FILE* output_file = fopen("output.txt", "w");
    if (output_file == NULL) {
        printf("Error opening output.txt\n");
        return 1;
    }

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < l; j++) {
            fprintf(output_file, "%d ", C[i * l + j]);
        }
        fprintf(output_file, "\n");
    }
    fclose(output_file);
    free(C);

    return 0;
}